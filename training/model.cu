#include "hip/hip_runtime.h"
#include <algorithm>
#include <cassert>
#include <iostream>
#include <thread>
#include <vector>
#include <unordered_map>
#include <hip/hip_runtime.h>
#include <hipblaslt.h>
#include "weights.h"

static const size_t ALIGNTO = 256;
static const size_t THREADS = 512;
static const size_t UNROLL = 8;
static const hipDataType DATA_TYPE = HIP_R_32F;
static const hipblasComputeType_t COMPUTE_TYPE = HIPBLAS_COMPUTE_32F_FAST_16F;

#define BLASOK(x) assert(x == HIPBLAS_STATUS_SUCCESS)
#define CUDAOK(x) assert(x == hipSuccess)
#define CHECK_KERNEL CUDAOK(hipGetLastError())
#define ALIGN(x) (ALIGNTO * (((x) / ALIGNTO) + ((x) % ALIGNTO != 0)))

#define DEVICE_ALLOC(device, count) {\
	if(device ## _capacity < count) {\
		CUDAOK(hipFree(device));\
		CUDAOK(hipMalloc(&device, count * sizeof(*device)));\
		device ## _capacity = count;\
	}\
}

#define PAGE_ALLOC(page, count) {\
	if(page ## _capacity < count) {\
		CUDAOK(hipHostFree(page));\
		CUDAOK(hipHostMalloc(&page, count * sizeof(*page)));\
		page ## _capacity = count;\
	}\
}

// global variables, not changed after init()
static hipblasLtHandle_t BLAS_HANDLE;
static hipblasLtMatmulDesc_t MM_DESC;
static hipblasLtMatrixLayout_t
	CONV_WEIGHT_DESC,
	HIDDEN_WEIGHT_DESC,
	OUTPUT_WEIGHT_DESC;
static hipblasLtMatmulPreference_t MM_PREFERENCE;
static float
	*EMBED_WEIGHTS = NULL,
	*OUT_WEIGHTS = NULL,
	*BACK_WEIGHTS = NULL,
	*HIDDEN_WEIGHTS = NULL,
	*HIDDEN_BIAS = NULL,
	*OUTPUT_WEIGHTS = NULL;

// thread-local global variables, changed once-per-cycle
static thread_local bool thread_initialised = false;
static thread_local uint8_t *p_upload, *d_upload;
static thread_local float *p_download, *d_download;
static thread_local int32_t
	num_nodes = 0,
	num_edges = 0,
	num_graphs = 0,
	*nodes = NULL,
	*batch = NULL;
static thread_local int2 *edges = NULL;
static thread_local float
	*forward_node_norm = NULL,
	*backward_node_norm = NULL,
	*graph_norms = NULL,
	*x = NULL,
	*out = NULL,
	*back = NULL,
	*scratch1 = NULL,
	*scratch2 = NULL,
	*pooled = NULL,
	*hidden = NULL;
static thread_local int32_t
	p_upload_capacity = 0,
	d_upload_capacity = 0,
	p_download_capacity = 0,
	d_download_capacity = 0,
	x_capacity = 0,
	out_capacity = 0,
	back_capacity = 0,
	scratch1_capacity = 0,
	scratch2_capacity = 0,
	pooled_capacity = 0,
	hidden_capacity = 0;
static thread_local hipblasLtMatrixLayout_t
	node_desc,
	pooled_desc,
	hidden_desc,
	output_desc;
static thread_local hipblasLtMatmulHeuristicResult_t
	conv_heuristic,
	hidden_heuristic,
	output_heuristic;

static void upload_weights(float **device, const float *data, int32_t size) {
	CUDAOK(hipMalloc(device, size));
	CUDAOK(hipMemcpy(
		*device,
		data,
		size,
		hipMemcpyHostToDevice
	));
}

static void init_matrix_layout(
	hipblasLtMatrixLayout_t layout,
	int32_t rows,
	int32_t cols
) {
	hipblasLtOrder_t order = HIPBLASLT_ORDER_ROW;
	BLASOK(cublasLtMatrixLayoutInit(
		layout,
		DATA_TYPE,
		rows,
		cols,
		cols
	));
	BLASOK(hipblasLtMatrixLayoutSetAttribute(
		layout,
		HIPBLASLT_MATRIX_LAYOUT_ORDER,
		&order,
		sizeof(order)
	));
}

static void mm(
	hipblasLtMatmulHeuristicResult_t *heuristic,
	hipblasLtMatrixLayout_t a_desc,
	float *a,
	hipblasLtMatrixLayout_t b_desc,
	float *b,
	hipblasLtMatrixLayout_t out_desc,
	float *out
) {
	float alpha = 1.0f;
	float beta = 0.0f;
	BLASOK(hipblasLtMatmul(
		BLAS_HANDLE,
		MM_DESC,
		&alpha,
		a,
		a_desc,
		b,
		b_desc,
		&beta,
		out,
		out_desc,
		out,
		out_desc,
		&heuristic->algo,
		NULL,
		0,
		hipStreamPerThread
	));
}

__global__ void k_embed(
	int32_t num_nodes,
	int32_t *nodes,
	float *weights,
	float *x
) {
	auto thread = threadIdx.x;
	auto channel = thread % CHANNELS;
	auto offset = thread / CHANNELS;

	#pragma unroll UNROLL
	for(int i = offset; i < num_nodes; i += THREADS / CHANNELS) {
		auto index = CHANNELS * i + channel;
		auto node = __ldcs(nodes + i);
		auto weight = __ldg(weights + CHANNELS * node + channel);
		__stcg(x + index, weight);
	}
}

static void embed() {
	k_embed<<<1, THREADS>>>(
		num_nodes,
		nodes,
		EMBED_WEIGHTS,
		x
	);
	CHECK_KERNEL;
}

__global__ void k_gather_neighbours(
	int32_t num_edges,
	float *x,
	int2 *edges,
	float *out,
	float *back
) {
	auto thread = threadIdx.x;
	auto channel = thread % CHANNELS;
	auto offset = thread / CHANNELS;

	#pragma unroll UNROLL
	for(int i = offset; i < num_edges; i += THREADS / CHANNELS) {
		auto edge = __ldg(edges + i);
		auto from = CHANNELS * edge.x + channel;
		auto to = CHANNELS * edge.y + channel;
		atomicAdd(out + to, __ldg(x + from));
		atomicAdd(back + from, __ldg(x + to));
	}
}

static void gather_neighbours() {
	CUDAOK(hipMemcpyAsync(
		out,
		x,
		num_nodes * CHANNELS * sizeof(float),
		hipMemcpyDeviceToDevice,
		hipStreamPerThread
	));
	CUDAOK(hipMemcpyAsync(
		back,
		x,
		num_nodes * CHANNELS * sizeof(float),
		hipMemcpyDeviceToDevice,
		hipStreamPerThread
	));
	k_gather_neighbours<<<1, THREADS>>>(
		num_edges,
		x,
		edges,
		out,
		back
	);
	CHECK_KERNEL;
}

__global__ void k_normalise(
	int32_t num_nodes,
	int2 *edges,
	float *forward_node_norm,
	float *backward_node_norm,
	float *out,
	float *back
) {
	auto thread = threadIdx.x;
	auto channel = thread % CHANNELS;
	auto offset = thread / CHANNELS;

	#pragma unroll UNROLL
	for(int i = offset; i < num_nodes; i += THREADS / CHANNELS) {
		auto index = CHANNELS * i + channel;
		float forward_norm = __ldg(forward_node_norm + i);
		float backward_norm = __ldg(backward_node_norm + i);
		float out_val = forward_norm * __ldcs(out + index);
		float back_val = backward_norm * __ldcs(back + index);
		__stcg(out + index, out_val);
		__stcg(back + index, back_val);
	}
}

static void normalise() {
	k_normalise<<<1, THREADS>>>(
		num_nodes,
		edges,
		forward_node_norm,
		backward_node_norm,
		out,
		back
	);
	CHECK_KERNEL;
}

__global__ void k_combine_scratch(
	int32_t num_nodes,
	float *scratch1,
	float *scratch2,
	float *x
) {
	auto thread = threadIdx.x;

	#pragma unroll UNROLL
	for(int i = thread; i < CHANNELS * num_nodes; i += THREADS) {
		float combined =
			fmaxf(0.0f, __ldg(scratch1 + i)) +
			fmaxf(0.0f, __ldg(scratch2 + i));
		atomicAdd(x + i, combined);
	}
}

static void combine_scratch() {
	k_combine_scratch<<<1, THREADS>>>(
		num_nodes,
		scratch1,
		scratch2,
		x
	);
	CHECK_KERNEL;
}

__global__ void k_global_mean_pool(
	int32_t num_nodes,
	int32_t num_graphs,
	int32_t *batch,
	float *graph_norms,
	float *x,
	float *pooled
) {
	auto thread = threadIdx.x;
	auto channel = thread % CHANNELS;
	auto offset = thread / CHANNELS;

	#pragma unroll UNROLL
	for(int i = offset; i < num_nodes; i += THREADS / CHANNELS) {
		float value = __ldcs(x + CHANNELS * i + channel);
		int32_t graph = __ldg(batch + i);
		float *addr = pooled + CHANNELS * graph + channel;
		atomicAdd(addr, value);
	}

	#pragma unroll UNROLL
	for(int i = offset; i < num_graphs; i += THREADS / CHANNELS) {
		float norm = __ldg(graph_norms + i);
		float *addr = pooled + CHANNELS * i + channel;
		float value = __ldcs(addr);
		__stcg(addr, norm * value);
	}
}

static void global_mean_pool() {
	CUDAOK(hipMemsetAsync(
		pooled,
		0,
		num_graphs * CHANNELS * sizeof(float),
		hipStreamPerThread
	));
	k_global_mean_pool<<<1, THREADS>>>(
		num_nodes,
		num_graphs,
		batch,
		graph_norms,
		x,
		pooled
	);
	CHECK_KERNEL;
}

__global__ void k_hidden_bias_relu(
	int32_t num_graphs,
	float *bias,
	float *hidden
) {
	auto thread = threadIdx.x;

	#pragma unroll UNROLL
	for(int i = thread; i < HIDDEN * num_graphs; i += THREADS) {
		float activated = fmaxf(
			0.0f,
			__ldcs(hidden + i) + __ldg(bias + i % HIDDEN)
		);
		__stcg(hidden + i, activated);
	}
}

static void hidden_bias_relu() {
	k_hidden_bias_relu<<<1, THREADS>>>(num_graphs, HIDDEN_BIAS, hidden);
	CHECK_KERNEL;
}

static void residual(int32_t layer) {
	float *out_weights = OUT_WEIGHTS + CHANNELS * CHANNELS * layer;
	float *back_weights = BACK_WEIGHTS + CHANNELS * CHANNELS * layer;

	gather_neighbours();
	normalise();
	mm(
		&conv_heuristic, 
		node_desc,
		out,
		CONV_WEIGHT_DESC,
		out_weights,
		node_desc,
		scratch1
	);
	mm(
		&conv_heuristic, 
		node_desc,
		back,
		CONV_WEIGHT_DESC,
		back_weights,
		node_desc,
		scratch2
	);
	combine_scratch();
}

static void upload(
	int32_t h_num_nodes,
	int32_t h_num_edges,
	int32_t h_num_graphs,
	const int32_t *h_nodes,
	const int32_t *h_sources,
	const int32_t *h_targets,
	const int32_t *h_batch
) {
	num_nodes = h_num_nodes;
	num_edges = h_num_edges;
	num_graphs = h_num_graphs;

	if(!thread_initialised) {
		BLASOK(hipblasLtMatrixLayoutCreate(
			&node_desc,
			DATA_TYPE,
			0,
			0,
			0
		));
		BLASOK(hipblasLtMatrixLayoutCreate(
			&pooled_desc,
			DATA_TYPE,
			0,
			0,
			0
		));
		BLASOK(hipblasLtMatrixLayoutCreate(
			&hidden_desc,
			DATA_TYPE,
			0,
			0,
			0
		));
		BLASOK(hipblasLtMatrixLayoutCreate(
			&output_desc,
			DATA_TYPE,
			0,
			0,
			0
		));
		thread_initialised = true;
	}
	init_matrix_layout(node_desc, num_nodes, CHANNELS);
	init_matrix_layout(pooled_desc, num_graphs, CHANNELS);
	init_matrix_layout(hidden_desc, num_graphs, 1024);
	init_matrix_layout(output_desc, num_graphs, 1);
	int _num_results;
	BLASOK(hipblasLtMatmulAlgoGetHeuristic(
		BLAS_HANDLE,
		MM_DESC,
		node_desc,
		CONV_WEIGHT_DESC,
		node_desc,
		node_desc,
		MM_PREFERENCE,
		1,
		&conv_heuristic,
		&_num_results
	));
	BLASOK(hipblasLtMatmulAlgoGetHeuristic(
		BLAS_HANDLE,
		MM_DESC,
		pooled_desc,
		HIDDEN_WEIGHT_DESC,
		hidden_desc,
		hidden_desc,
		MM_PREFERENCE,
		1,
		&hidden_heuristic,
		&_num_results
	));
	BLASOK(hipblasLtMatmulAlgoGetHeuristic(
		BLAS_HANDLE,
		MM_DESC,
		hidden_desc,
		OUTPUT_WEIGHT_DESC,
		output_desc,
		output_desc,
		MM_PREFERENCE,
		1,
		&output_heuristic,
		&_num_results
	));

	size_t node_offset = 0;
	auto edge_offset = ALIGN(
		node_offset + num_nodes * sizeof(int32_t)
	);
	auto forward_node_norm_offset = ALIGN(
		edge_offset + num_edges * sizeof(int2)
	);
	auto backward_node_norm_offset = ALIGN(
		forward_node_norm_offset + num_nodes * sizeof(float)
	);
	auto batch_offset = ALIGN(
		backward_node_norm_offset + num_nodes * sizeof(float)
	);
	auto graph_norm_offset = ALIGN(
		batch_offset + num_nodes * sizeof(int32_t)
	);
	auto upload_bytes = graph_norm_offset + num_graphs * sizeof(float);
	DEVICE_ALLOC(d_upload, upload_bytes);
	DEVICE_ALLOC(d_download, num_graphs);
	DEVICE_ALLOC(x, num_nodes * CHANNELS);
	DEVICE_ALLOC(out, num_nodes * CHANNELS);
	DEVICE_ALLOC(back, num_nodes * CHANNELS);
	DEVICE_ALLOC(scratch1, num_nodes * CHANNELS);
	DEVICE_ALLOC(scratch2, num_nodes * CHANNELS);
	DEVICE_ALLOC(pooled, num_graphs * CHANNELS);
	DEVICE_ALLOC(hidden, num_graphs * HIDDEN);
	PAGE_ALLOC(p_upload, upload_bytes);
	PAGE_ALLOC(p_download, num_graphs);

	nodes = (int32_t *)(d_upload + node_offset);
	edges = (int2 *)(d_upload + edge_offset);
	forward_node_norm = (float *)(d_upload + forward_node_norm_offset);
	backward_node_norm = (float *)(d_upload + backward_node_norm_offset);
	batch = (int32_t *)(d_upload + batch_offset);
	graph_norms = (float *)(d_upload + graph_norm_offset);

	// alignment-safe: should be aligned from before
	auto p_nodes = (int32_t *)(p_upload + node_offset);
	auto p_edges = (int2 *)(p_upload + edge_offset);
	auto p_forward_node_norm =
		(float *)(p_upload + forward_node_norm_offset);
	auto p_backward_node_norm =
		(float *)(p_upload + backward_node_norm_offset);
	auto p_batch = (int32_t *)(p_upload + batch_offset);
	auto p_graph_norms = (float *)(p_upload + graph_norm_offset);

	memcpy(p_batch, h_batch, num_nodes * sizeof(int32_t));
	memset(p_forward_node_norm, 0, num_nodes * sizeof(float));
	memset(p_backward_node_norm, 0, num_nodes * sizeof(float));
	memset(p_graph_norms, 0, num_graphs * sizeof(float));
	for(int i = 0; i < num_edges; i++) {
		auto source = h_sources[i];
		auto target = h_targets[i];
		p_edges[i].x = source;
		p_edges[i].y = target;
		p_forward_node_norm[target] += 1.0f;
		p_backward_node_norm[source] += 1.0f;
	}
	for(int i = 0; i < num_nodes; i++) {
		p_nodes[i] = h_nodes[i];
		p_forward_node_norm[i] = 1.0f /
			(1.0f + p_forward_node_norm[i]);
		p_backward_node_norm[i] = 1.0f /
			(1.0f + p_backward_node_norm[i]);
		p_graph_norms[h_batch[i]] += 1.0f;
	}
	for(int i = 0; i < num_graphs; i++) {
		p_graph_norms[i] = 1.0f / p_graph_norms[i];
	}

	CUDAOK(hipMemcpyAsync(
		d_upload,
		p_upload,
		upload_bytes,
		hipMemcpyHostToDevice,
		hipStreamPerThread
	));
}

static void download(float *h_results) {
	CUDAOK(hipMemcpyAsync(
		p_download,
		d_download,
		num_graphs * sizeof(float),
		hipMemcpyDeviceToHost,
		hipStreamPerThread
	));
	CUDAOK(hipStreamSynchronize(hipStreamPerThread));

	for(int i = 0; i < num_graphs; i++) {
		h_results[i] = p_download[i] + OUTPUT_BIAS;
	}
}

extern "C" void init() {
	BLASOK(hipblasLtCreate(&BLAS_HANDLE));
	BLASOK(hipblasLtMatmulPreferenceCreate(&MM_PREFERENCE));
	BLASOK(hipblasLtMatmulDescCreate(&MM_DESC, COMPUTE_TYPE, DATA_TYPE));
	BLASOK(hipblasLtMatrixLayoutCreate(
		&CONV_WEIGHT_DESC,
		DATA_TYPE,
		0,
		0,
		0
	));
	init_matrix_layout(CONV_WEIGHT_DESC, CHANNELS, CHANNELS);
	BLASOK(hipblasLtMatrixLayoutCreate(
		&HIDDEN_WEIGHT_DESC,
		DATA_TYPE,
		0,
		0,
		0
	));
	init_matrix_layout(HIDDEN_WEIGHT_DESC, CHANNELS, HIDDEN);
	BLASOK(hipblasLtMatrixLayoutCreate(
		&OUTPUT_WEIGHT_DESC,
		DATA_TYPE,
		0,
		0,
		0
	));
	init_matrix_layout(OUTPUT_WEIGHT_DESC, HIDDEN, 1);

	upload_weights(
		&EMBED_WEIGHTS,
		EMBED_WEIGHTS_DATA,
		sizeof(EMBED_WEIGHTS_DATA)
	);
	upload_weights(
		&OUT_WEIGHTS,
		OUT_WEIGHTS_DATA,
		sizeof(OUT_WEIGHTS_DATA)
	);
	upload_weights(
		&BACK_WEIGHTS,
		BACK_WEIGHTS_DATA,
		sizeof(BACK_WEIGHTS_DATA)
	);
	upload_weights(
		&HIDDEN_WEIGHTS,
		HIDDEN_WEIGHT_DATA,
		sizeof(HIDDEN_WEIGHT_DATA)
	);
	upload_weights(
		&HIDDEN_BIAS,
		HIDDEN_BIAS_DATA,
		sizeof(HIDDEN_BIAS_DATA)
	);
	upload_weights(
		&OUTPUT_WEIGHTS,
		OUTPUT_WEIGHT_DATA,
		sizeof(OUTPUT_WEIGHT_DATA)
	);
}

extern "C" void model(
	int32_t h_num_nodes,
	int32_t h_num_edges,
	int32_t h_num_graphs,
	const int32_t *h_nodes,
	const int32_t *h_sources,
	const int32_t *h_targets,
	const int32_t *h_batch,
	float  *h_results
) {
	upload(
		h_num_nodes,
		h_num_edges,
		h_num_graphs,
		h_nodes,
		h_sources,
		h_targets,
		h_batch
	);
	embed();
	for(int i = 0; i < LAYERS; i++) {
		residual(i);
	}
	global_mean_pool();
	mm(
		&hidden_heuristic,
		pooled_desc,
		pooled,
		HIDDEN_WEIGHT_DESC,
		HIDDEN_WEIGHTS,
		hidden_desc,
		hidden
	);
	hidden_bias_relu();
	mm(
		&output_heuristic,
		hidden_desc,
		hidden,
		OUTPUT_WEIGHT_DESC,
		OUTPUT_WEIGHTS,
		output_desc,
		d_download
	);
	download(h_results);
}

#ifndef NO_TEST
static void go() {
	const int num_nodes = 1000;
	const int num_edges = 2000;
	const int num_graphs = 10;
	int32_t nodes[num_nodes];
	int32_t sources[num_edges];
	int32_t targets[num_edges];
	int32_t batch[num_nodes];
	for(int i = 0; i < num_nodes; i++) {
		nodes[i] = 0;
		batch[i] = i / (num_nodes / num_graphs);
	}
	for(int i = 0; i < num_edges; i++) {
		sources[i] = i / 2;
		targets[i] = i / 2;
	}

	float output[num_graphs];
	model(
		num_nodes,
		num_edges,
		num_graphs,
		nodes,
		sources,
		targets,
		batch,
		output
	);
	std::cout << output[0] << std::endl;
}

int main() {
	init();

	std::vector<std::thread> workers;
	auto f = []() {
		for(int i = 0; i < 50; i++) {
			go();
		}
	};
	for(int i = 0; i < 8; i++) {
		workers.emplace_back(f);
	}
	for(auto &worker : workers) {
		worker.join();
	}
}
#endif
