#include "hip/hip_runtime.h"
#include <cassert>
#include <hip/hip_runtime.h>
#include <hipblaslt.h>
#include "weights.h"

static const size_t
	ALIGNTO = 256,
	BLOCKS = 32,
	UNROLL = 8;
static const hipDataType DATA_TYPE = HIP_R_32F;
static const hipblasComputeType_t COMPUTE_TYPE = HIPBLAS_COMPUTE_32F;

#define BLASOK(x) assert(x == HIPBLAS_STATUS_SUCCESS)
#define CUDAOK(x) assert(x == hipSuccess)
#define ALIGN(x) (ALIGNTO * (((x) / ALIGNTO) + ((x) % ALIGNTO != 0)))

#define DEVICE_ALLOC(device, count) {\
	if(device ## _capacity < count) {\
		CUDAOK(hipFree(device));\
		CUDAOK(hipMalloc(&device, count * sizeof(*device)));\
		device ## _capacity = count;\
	}\
}

#define PAGE_ALLOC(page, count) {\
	if(page ## _capacity < count) {\
		CUDAOK(hipHostFree(page));\
		CUDAOK(hipHostMalloc(&page, count * sizeof(*page)));\
		page ## _capacity = count;\
	}\
}

// global variables, not changed after init()
static hipblasLtHandle_t BLAS_HANDLE;
static hipblasLtMatmulDesc_t MM_DESC;
static hipblasLtMatrixLayout_t
	CONV_WEIGHT_DESC,
	HIDDEN_WEIGHT_DESC,
	OUTPUT_WEIGHT_DESC;
static hipblasLtMatmulPreference_t MM_PREFERENCE;
static float
	*EMBED_WEIGHTS = NULL,
	*OUT_WEIGHTS = NULL,
	*BACK_WEIGHTS = NULL,
	*OUT_BIAS = NULL,
	*BACK_BIAS = NULL,
	*HIDDEN_WEIGHTS = NULL,
	*HIDDEN_BIAS = NULL,
	*OUTPUT_WEIGHTS = NULL;

// thread-local global variables, changed once-per-cycle
static thread_local bool thread_initialised = false;
static thread_local uint8_t *p_upload, *d_upload;
static thread_local float *p_download, *d_download;
static thread_local int32_t *nodes = NULL, *batch = NULL;
static thread_local int2 *edges = NULL;
static thread_local float
	*forward_node_norm = NULL,
	*backward_node_norm = NULL,
	*graph_norm = NULL,
	*x = NULL,
	*out = NULL,
	*back = NULL,
	*out_scratch = NULL,
	*back_scratch = NULL,
	*pooled = NULL,
	*hidden = NULL;
static thread_local uint32_t
	num_nodes = 0,
	num_edges = 0,
	num_graphs = 0,
	p_upload_capacity = 0,
	d_upload_capacity = 0,
	p_download_capacity = 0,
	d_download_capacity = 0,
	x_capacity = 0,
	out_capacity = 0,
	back_capacity = 0,
	out_scratch_capacity = 0,
	back_scratch_capacity = 0,
	pooled_capacity = 0,
	hidden_capacity = 0;
static thread_local hipblasLtMatrixLayout_t
	node_desc,
	pooled_desc,
	hidden_desc,
	output_desc;
static thread_local hipblasLtMatmulHeuristicResult_t
	conv_heuristic,
	hidden_heuristic,
	output_heuristic;

static void upload_weights(float **device, const float *data, size_t size) {
	CUDAOK(hipMalloc(device, size));
	CUDAOK(hipMemcpy(
		*device,
		data,
		size,
		hipMemcpyHostToDevice
	));
}

static void init_matrix_layout(
	hipblasLtMatrixLayout_t layout,
	int32_t rows,
	int32_t cols
) {
	hipblasLtOrder_t order = HIPBLASLT_ORDER_ROW;
	BLASOK(cublasLtMatrixLayoutInit(
		layout,
		DATA_TYPE,
		rows,
		cols,
		cols
	));
	BLASOK(hipblasLtMatrixLayoutSetAttribute(
		layout,
		HIPBLASLT_MATRIX_LAYOUT_ORDER,
		&order,
		sizeof(order)
	));
}

static void mm(
	hipblasLtMatmulHeuristicResult_t *heuristic,
	hipblasLtMatrixLayout_t a_desc,
	float *a,
	hipblasLtMatrixLayout_t b_desc,
	float *b,
	hipblasLtMatrixLayout_t out_desc,
	float *out
) {
	float alpha = 1.0f;
	float beta = 0.0f;
	BLASOK(hipblasLtMatmul(
		BLAS_HANDLE,
		MM_DESC,
		&alpha,
		a,
		a_desc,
		b,
		b_desc,
		&beta,
		out,
		out_desc,
		out,
		out_desc,
		&heuristic->algo,
		NULL,
		0,
		hipStreamPerThread
	));
}

__global__ void k_embed(
	int32_t num_nodes,
	int32_t *nodes,
	float *weights,
	float *x
) {
	auto offset = blockIdx.x;
	auto channel = threadIdx.x;

	#pragma unroll UNROLL
	for(int i = offset; i < num_nodes; i += BLOCKS) {
		auto index = CHANNELS * i + channel;
		auto node = __ldg(nodes + i);
		auto weight = __ldg(weights + CHANNELS * node + channel);
		__stcg(x + index, weight);
	}
}

static void embed() {
	k_embed<<<BLOCKS, CHANNELS>>>(
		num_nodes,
		nodes,
		EMBED_WEIGHTS,
		x
	);
}

__global__ void k_gather_neighbours(
	uint32_t num_edges,
	float *x,
	int2 *edges,
	float *out,
	float *back
) {
	auto offset = blockIdx.x;
	auto channel = threadIdx.x;

	#pragma unroll UNROLL
	for(int i = offset; i < num_edges; i += BLOCKS) {
		auto edge = __ldg(edges + i);
		auto from = CHANNELS * edge.x + channel;
		auto to = CHANNELS * edge.y + channel;
		atomicAdd(out + to, __ldg(x + from));
		atomicAdd(back + from, __ldg(x + to));
	}
}

static void gather_neighbours() {
	CUDAOK(hipMemcpyAsync(
		out,
		x,
		num_nodes * CHANNELS * sizeof(float),
		hipMemcpyDeviceToDevice,
		hipStreamPerThread
	));
	CUDAOK(hipMemcpyAsync(
		back,
		x,
		num_nodes * CHANNELS * sizeof(float),
		hipMemcpyDeviceToDevice,
		hipStreamPerThread
	));
	k_gather_neighbours<<<BLOCKS, CHANNELS>>>(
		num_edges,
		x,
		edges,
		out,
		back
	);
}

__global__ void k_normalise(
	int32_t num_nodes,
	float *forward_node_norm,
	float *backward_node_norm,
	float *out,
	float *back
) {
	auto offset = blockIdx.x;
	auto channel = threadIdx.x;

	#pragma unroll UNROLL
	for(int i = offset; i < num_nodes; i += BLOCKS) {
		auto index = CHANNELS * i + channel;
		float forward_norm = __ldg(forward_node_norm + i);
		float backward_norm = __ldg(backward_node_norm + i);
		float out_val = forward_norm * __ldg(out + index);
		float back_val = backward_norm * __ldg(back + index);
		__stcg(out + index, out_val);
		__stcg(back + index, back_val);
	}
}

static void normalise() {
	k_normalise<<<BLOCKS, CHANNELS>>>(
		num_nodes,
		forward_node_norm,
		backward_node_norm,
		out,
		back
	);
}

__global__ void k_bias_relu_sum(
	int32_t num_nodes,
	float *out_bias,
	float *back_bias,
	float *out_scratch,
	float *back_scratch,
	float *x
) {
	auto offset = blockIdx.x;
	auto channel = threadIdx.x;
	float outb = __ldg(out_bias + channel);
	float backb = __ldg(back_bias + channel);

	#pragma unroll UNROLL
	for(int i = offset; i < num_nodes; i += BLOCKS) {
		auto index = CHANNELS * i + channel;
		float current = __ldg(x + index);
		float combined = current +
			fmaxf(0.0f, outb + __ldg(out_scratch + index)) +
			fmaxf(0.0f, backb + __ldg(back_scratch + index));
		__stcg(x + index, combined);
	}
}

static void bias_relu_sum(float *out_bias, float *back_bias) {
	k_bias_relu_sum<<<BLOCKS, CHANNELS>>>(
		num_nodes,
		out_bias,
		back_bias,
		out_scratch,
		back_scratch,
		x
	);
}

__global__ void k_global_mean_pool(
	int32_t num_nodes,
	int32_t num_graphs,
	int32_t *batch,
	float *x,
	float *graph_norm,
	float *pooled
) {
	auto offset = blockIdx.x;
	auto channel = threadIdx.x;

	#pragma unroll UNROLL
	for(int i = offset; i < num_nodes; i += BLOCKS) {
		auto graph = __ldg(batch + i);
		auto norm = __ldg(graph_norm + graph);
		auto value = __ldg(x + CHANNELS * i + channel);
		atomicAdd(pooled + CHANNELS * graph + channel, norm * value);
	}
}

static void global_mean_pool() {
	k_global_mean_pool<<<BLOCKS, CHANNELS>>>(
		num_nodes,
		num_graphs,
		batch,
		x,
		graph_norm,
		pooled
	);
}

__global__ void k_hidden_bias_relu(
	int32_t num_graphs,
	float *bias,
	float *hidden
) {
	auto channel = threadIdx.x;
	#pragma unroll UNROLL
	for(int i = 0; i < num_graphs; i++) {
		auto index = HIDDEN * i + channel;
		float activated = fmaxf(
			0.0f,
			__ldg(hidden + index) + __ldg(bias + channel)
		);
		__stcg(hidden + index, activated);
	}
}

static void hidden_bias_relu() {
	k_hidden_bias_relu<<<1, HIDDEN>>>(num_graphs, HIDDEN_BIAS, hidden);
}

static void residual(int32_t layer) {
	float *out_weights = OUT_WEIGHTS + CHANNELS * CHANNELS * layer;
	float *back_weights = BACK_WEIGHTS + CHANNELS * CHANNELS * layer;
	float *out_bias = OUT_BIAS + CHANNELS * layer;
	float *back_bias = BACK_BIAS + CHANNELS * layer;

	gather_neighbours();
	normalise();
	mm(
		&conv_heuristic, 
		node_desc,
		out,
		CONV_WEIGHT_DESC,
		out_weights,
		node_desc,
		out_scratch
	);
	mm(
		&conv_heuristic, 
		node_desc,
		back,
		CONV_WEIGHT_DESC,
		back_weights,
		node_desc,
		back_scratch
	);
	bias_relu_sum(out_bias, back_bias);
}

static void upload(
	uint32_t h_num_nodes,
	uint32_t h_num_edges,
	uint32_t h_num_graphs,
	const uint32_t *h_nodes,
	const uint32_t *h_sources,
	const uint32_t *h_targets,
	const uint32_t *h_batch
) {
	num_nodes = h_num_nodes;
	num_edges = h_num_edges;
	num_graphs = h_num_graphs;

	if(!thread_initialised) {
		BLASOK(hipblasLtMatrixLayoutCreate(
			&node_desc,
			DATA_TYPE,
			0,
			0,
			0
		));
		BLASOK(hipblasLtMatrixLayoutCreate(
			&pooled_desc,
			DATA_TYPE,
			0,
			0,
			0
		));
		BLASOK(hipblasLtMatrixLayoutCreate(
			&hidden_desc,
			DATA_TYPE,
			0,
			0,
			0
		));
		BLASOK(hipblasLtMatrixLayoutCreate(
			&output_desc,
			DATA_TYPE,
			0,
			0,
			0
		));
		thread_initialised = true;
	}
	init_matrix_layout(node_desc, num_nodes, CHANNELS);
	init_matrix_layout(pooled_desc, num_graphs, CHANNELS);
	init_matrix_layout(hidden_desc, num_graphs, HIDDEN);
	init_matrix_layout(output_desc, num_graphs, 1);
	int _num_results;
	BLASOK(hipblasLtMatmulAlgoGetHeuristic(
		BLAS_HANDLE,
		MM_DESC,
		node_desc,
		CONV_WEIGHT_DESC,
		node_desc,
		node_desc,
		MM_PREFERENCE,
		1,
		&conv_heuristic,
		&_num_results
	));
	BLASOK(hipblasLtMatmulAlgoGetHeuristic(
		BLAS_HANDLE,
		MM_DESC,
		pooled_desc,
		HIDDEN_WEIGHT_DESC,
		hidden_desc,
		hidden_desc,
		MM_PREFERENCE,
		1,
		&hidden_heuristic,
		&_num_results
	));
	BLASOK(hipblasLtMatmulAlgoGetHeuristic(
		BLAS_HANDLE,
		MM_DESC,
		hidden_desc,
		OUTPUT_WEIGHT_DESC,
		output_desc,
		output_desc,
		MM_PREFERENCE,
		1,
		&output_heuristic,
		&_num_results
	));

	size_t node_offset = 0;
	auto edge_offset = ALIGN(
		node_offset + num_nodes * sizeof(int32_t)
	);
	auto forward_node_norm_offset = ALIGN(
		edge_offset + num_edges * sizeof(int2)
	);
	auto backward_node_norm_offset = ALIGN(
		forward_node_norm_offset + num_nodes * sizeof(float)
	);
	auto graph_norm_offset = ALIGN(
		backward_node_norm_offset + num_nodes * sizeof(float)
	);
	auto batch_offset = ALIGN(
		graph_norm_offset + num_graphs * sizeof(float)
	);
	auto upload_bytes = ALIGN(
		batch_offset + num_nodes * sizeof(int32_t)
	);

	DEVICE_ALLOC(d_upload, upload_bytes);
	DEVICE_ALLOC(d_download, num_graphs);
	DEVICE_ALLOC(x, num_nodes * CHANNELS);
	DEVICE_ALLOC(out, num_nodes * CHANNELS);
	DEVICE_ALLOC(back, num_nodes * CHANNELS);
	DEVICE_ALLOC(out_scratch, num_nodes * CHANNELS);
	DEVICE_ALLOC(back_scratch, num_nodes * CHANNELS);
	DEVICE_ALLOC(pooled, num_graphs * CHANNELS);
	DEVICE_ALLOC(hidden, num_graphs * HIDDEN);
	PAGE_ALLOC(p_upload, upload_bytes);
	PAGE_ALLOC(p_download, num_graphs);
	CUDAOK(hipMemsetAsync(
		pooled,
		0,
		num_graphs * CHANNELS * sizeof(float),
		hipStreamPerThread
	));

	// alignment-safe: should be aligned from before
	auto p_nodes = (int32_t *)(p_upload + node_offset);
	auto p_edges = (int2 *)(p_upload + edge_offset);
	auto p_forward_node_norm =
		(float *)(p_upload + forward_node_norm_offset);
	auto p_backward_node_norm =
		(float *)(p_upload + backward_node_norm_offset);
	auto p_graph_norm = (float *)(p_upload + graph_norm_offset);
	auto p_batch = (int32_t *)(p_upload + batch_offset);

	memset(p_forward_node_norm, 0, num_nodes * sizeof(float));
	memset(p_backward_node_norm, 0, num_nodes * sizeof(float));
	memset(p_graph_norm, 0, num_graphs * sizeof(float));
	for(uint32_t i = 0; i < num_edges; i++) {
		auto source = h_sources[i];
		auto target = h_targets[i];
		p_edges[i].x = source;
		p_edges[i].y = target;
		p_forward_node_norm[target] += 1.0f;
		p_backward_node_norm[source] += 1.0f;
	}
	for(uint32_t i = 0; i < num_nodes; i++) {
		p_nodes[i] = h_nodes[i];
		p_batch[i] = h_batch[i];
		p_graph_norm[p_batch[i]] += 1.0;
		p_forward_node_norm[i] = 1.0f /
			(1.0f + p_forward_node_norm[i]);
		p_backward_node_norm[i] = 1.0f /
			(1.0f + p_backward_node_norm[i]);
	}
	for(uint32_t i = 0; i < num_graphs; i++) {
		p_graph_norm[i] = 1.0f / p_graph_norm[i];
	}

	CUDAOK(hipMemcpyAsync(
		d_upload,
		p_upload,
		upload_bytes,
		hipMemcpyHostToDevice,
		hipStreamPerThread
	));

	nodes = (int32_t *)(d_upload + node_offset);
	edges = (int2 *)(d_upload + edge_offset);
	forward_node_norm = (float *)(d_upload + forward_node_norm_offset);
	backward_node_norm = (float *)(d_upload + backward_node_norm_offset);
	graph_norm = (float *)(d_upload + graph_norm_offset);
	batch = (int32_t *)(d_upload + batch_offset);
}

static void download(float *h_results) {
	CUDAOK(hipMemcpyAsync(
		p_download,
		d_download,
		num_graphs * sizeof(float),
		hipMemcpyDeviceToHost,
		hipStreamPerThread
	));
	CUDAOK(hipStreamSynchronize(hipStreamPerThread));

	for(uint32_t i = 0; i < num_graphs; i++) {
		h_results[i] = p_download[i];
	}
}

extern "C" void init() {
	BLASOK(hipblasLtCreate(&BLAS_HANDLE));
	BLASOK(hipblasLtMatmulPreferenceCreate(&MM_PREFERENCE));
	BLASOK(hipblasLtMatmulDescCreate(&MM_DESC, COMPUTE_TYPE, DATA_TYPE));
	BLASOK(hipblasLtMatrixLayoutCreate(
		&CONV_WEIGHT_DESC,
		DATA_TYPE,
		0,
		0,
		0
	));
	init_matrix_layout(CONV_WEIGHT_DESC, CHANNELS, CHANNELS);
	BLASOK(hipblasLtMatrixLayoutCreate(
		&HIDDEN_WEIGHT_DESC,
		DATA_TYPE,
		0,
		0,
		0
	));
	init_matrix_layout(HIDDEN_WEIGHT_DESC, CHANNELS, HIDDEN);
	BLASOK(hipblasLtMatrixLayoutCreate(
		&OUTPUT_WEIGHT_DESC,
		DATA_TYPE,
		0,
		0,
		0
	));
	init_matrix_layout(OUTPUT_WEIGHT_DESC, HIDDEN, 1);

	upload_weights(
		&EMBED_WEIGHTS,
		EMBED_WEIGHTS_DATA,
		sizeof(EMBED_WEIGHTS_DATA)
	);
	upload_weights(
		&OUT_WEIGHTS,
		OUT_WEIGHTS_DATA,
		sizeof(OUT_WEIGHTS_DATA)
	);
	upload_weights(
		&BACK_WEIGHTS,
		BACK_WEIGHTS_DATA,
		sizeof(BACK_WEIGHTS_DATA)
	);
	upload_weights(
		&OUT_BIAS,
		OUT_BIAS_DATA,
		sizeof(OUT_BIAS_DATA)
	);
	upload_weights(
		&BACK_BIAS,
		BACK_BIAS_DATA,
		sizeof(BACK_BIAS_DATA)
	);
	upload_weights(
		&HIDDEN_WEIGHTS,
		HIDDEN_WEIGHT_DATA,
		sizeof(HIDDEN_WEIGHT_DATA)
	);
	upload_weights(
		&HIDDEN_BIAS,
		HIDDEN_BIAS_DATA,
		sizeof(HIDDEN_BIAS_DATA)
	);
	upload_weights(
		&OUTPUT_WEIGHTS,
		OUTPUT_WEIGHT_DATA,
		sizeof(OUTPUT_WEIGHT_DATA)
	);
}

#include <cstdio>
extern "C" void model(
	uint32_t h_num_nodes,
	uint32_t h_num_edges,
	uint32_t h_num_graphs,
	const uint32_t *h_nodes,
	const uint32_t *h_sources,
	const uint32_t *h_targets,
	const uint32_t *h_batch,
	float *h_results
) {
	upload(
		h_num_nodes,
		h_num_edges,
		h_num_graphs,
		h_nodes,
		h_sources,
		h_targets,
		h_batch
	);
	embed();
	for(uint32_t i = 0; i < LAYERS; i++) {
		residual(i);
	}
	global_mean_pool();
	mm(
		&hidden_heuristic,
		pooled_desc,
		pooled,
		HIDDEN_WEIGHT_DESC,
		HIDDEN_WEIGHTS,
		hidden_desc,
		hidden
	);
	hidden_bias_relu();
	mm(
		&output_heuristic,
		hidden_desc,
		hidden,
		OUTPUT_WEIGHT_DESC,
		OUTPUT_WEIGHTS,
		output_desc,
		d_download
	);
	download(h_results);
}
